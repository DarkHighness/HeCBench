/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.

 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions
 are met:
  * Redistributions of source code must retain the above copyright
    notice, this list of conditions and the following disclaimer.
  * Redistributions in binary form must reproduce the above copyright
    notice, this list of conditions and the following disclaimer in the
    documentation and/or other materials provided with the distribution.
  * Neither the name of NVIDIA CORPORATION nor the names of its
    contributors may be used to endorse or promote products derived
    from this software without specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

long long get_time() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (tv.tv_sec * 1000000) + tv.tv_usec;
}

/* Add two vectors on the GPU */
__global__ void vectorAddGPU(float *__restrict__ a,
                             float *__restrict__ b,
                             float *__restrict__ c,
                             int N) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    c[idx] = a[idx] + b[idx];
  }
}

// Macro to aligned up to the memory size in question
#define MEMORY_ALIGNMENT 4096
#define ALIGN_UP(x, size) (((size_t)x + (size - 1)) & (~(size - 1)))

void eval (bool bPinGenericMemory) {
  int n, nelem;
  unsigned int flags;
  size_t bytes;
  float *a, *b, *c;           // Pinned memory allocated on the CPU
  float *a_UA, *b_UA, *c_UA;  // Non-4K Aligned Pinned memory on the CPU
  float *d_a, *d_b, *d_c;     // Device pointers for mapped memory
  float errorNorm, refNorm, ref, diff;


#if defined(__APPLE__) || defined(MACOSX)
    bPinGenericMemory = false;
    printf("Warning: Generic Pinning of System Paged memory is not support on MacOS\n");
#endif

  if (bPinGenericMemory) {
    printf("> Using Generic System Paged Memory (malloc)\n");
  } else {
    printf("> Using Host Allocated (cudaHostAlloc)\n");
  }

  /* Allocate mapped CPU memory. */

  for (nelem = 1024*1024; nelem <= (1024*1024*128); nelem = nelem*2) {
    bytes = nelem * sizeof(float);

    auto start = get_time();

    if (bPinGenericMemory) {
      // Allocate generic memory with malloc() and pin it later 
      // instead of using cudaHostAlloc()
      a_UA = (float *)malloc(bytes + MEMORY_ALIGNMENT);
      b_UA = (float *)malloc(bytes + MEMORY_ALIGNMENT);
      c_UA = (float *)malloc(bytes + MEMORY_ALIGNMENT);

      // We need to ensure memory is aligned to 4K (so we will need to padd memory
      // accordingly)
      a = (float *)ALIGN_UP(a_UA, MEMORY_ALIGNMENT);
      b = (float *)ALIGN_UP(b_UA, MEMORY_ALIGNMENT);
      c = (float *)ALIGN_UP(c_UA, MEMORY_ALIGNMENT);

      hipHostRegister(a, bytes, hipHostRegisterMapped);
      hipHostRegister(b, bytes, hipHostRegisterMapped);
      hipHostRegister(c, bytes, hipHostRegisterMapped);
    } else {
      flags = hipHostMallocMapped;
      hipHostAlloc((void **)&a, bytes, flags);
      hipHostAlloc((void **)&b, bytes, flags);
      hipHostAlloc((void **)&c, bytes, flags);
    }

    /* Initialize the vectors. */
    for (n = 0; n < nelem; n++) {
      a[n] = rand() / (float)RAND_MAX;
      b[n] = rand() / (float)RAND_MAX;
    }

    /* Get the device pointers for the pinned CPU memory mapped into the GPU
       memory space. */
    hipHostGetDevicePointer((void **)&d_a, (void *)a, 0);
    hipHostGetDevicePointer((void **)&d_b, (void *)b, 0);
    hipHostGetDevicePointer((void **)&d_c, (void *)c, 0);

    /* Call the GPU kernel using the pointers residing in CPU mapped memory.
     */
    dim3 block(256);
    dim3 grid((unsigned int)ceil(nelem / (float)block.x));
    vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, nelem);
    hipDeviceSynchronize();

    /* Compare the results */

    errorNorm = 0.f;
    refNorm = 0.f;

    for (n = 0; n < nelem; n++) {
      ref = a[n] + b[n];
      diff = c[n] - ref;
      errorNorm += diff * diff;
      refNorm += ref * ref;
    }

    errorNorm = (float)sqrt((double)errorNorm);
    refNorm = (float)sqrt((double)refNorm);

    printf("%s ", (errorNorm / refNorm < 1.e-6f) ? "SUCCESS" : "FAILURE");

    /* Memory clean up */


    if (bPinGenericMemory) {
      hipHostUnregister(a);
      hipHostUnregister(b);
      hipHostUnregister(c);
      free(a_UA);
      free(b_UA);
      free(c_UA);
    } else {
      hipHostFree(a);
      hipHostFree(b);
      hipHostFree(c);
    }

    auto end = get_time();
    printf("Total elapsed time: %.2f s\n", (end - start) / 1e6f);
  }
}

int main(int argc, char **argv) {
  bool bPinGenericMemory;

  bPinGenericMemory = false;
  eval(bPinGenericMemory); 

  bPinGenericMemory = true;
  eval(bPinGenericMemory); 
  return 0;
}
